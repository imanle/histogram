#include "hip/hip_runtime.h"

#include "common.h"
#include "timer.h"

__global__ void histogram_private_kernel(unsigned char* image, unsigned int* bins, unsigned int width, unsigned int height) {
     __shared__ int hist_s [NUM_BINS];
     unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
     if(i<NUM_BINS){
          hist_s[i]=0;
     }
    __synchthreads();
     
    if(i < width * height) {
        unsigned char b = image[i];
        atomicAdd(&hist_s[b], 1);
    }
     __syncthreads();
     
    if (hist_s[threadIdx.x] > 0 && i < NUM_BINS) {
        unsigned char b = image[i];
        atomicAdd(&bins[b],hist_s[b]);
    }
}

void histogram_gpu_private(unsigned char* image_d, unsigned int* bins_d, unsigned int width, unsigned int height) {

     const unsigned int numThreadsPerBlock = 1024;
     const unsigned int numBlocks = (width * height + numThreadsPerBlock - 1)/numThreadsPerBlock;
     histogram_private_kernel <<< numThreadsPerBlock, numBlocks >>>(image_d,bins_d, width,height);
}

__global__ void histogram_private_coarse_kernel(unsigned char* image, unsigned int* bins, unsigned int width, unsigned int height) {

    // TODO














}

void histogram_gpu_private_coarse(unsigned char* image_d, unsigned int* bins_d, unsigned int width, unsigned int height) {

    // TODO





}

